#include "hip/hip_runtime.h"
//
// Created by ADMIN on 2024/3/8.
//

#include "dfsph_cuda_api.cuh"

namespace SoSim {

    __device__ inline float
    cubic_value(const Vec3f &r, float h) {
        float r_norm = r.length();
        const float PI = 3.14159265;
        const float cubicSigma = 8.f / PI / static_cast<float>(std::pow(h, 3));

        float res = 0.0;
        float invH = 1 / h;
        float q = r_norm * invH;

        if (q <= 1) {
            if (q <= 0.5) {
                auto q2 = q * q;
                auto q3 = q2 * q;
                res = static_cast<float>(cubicSigma * (6.0 * q3 - 6.0 * q2 + 1));
            } else {
                res = static_cast<float>(cubicSigma * 2 * std::pow(1 - q, 3));
            }
        }

        return res;
    }

    __device__ inline Vec3f
    cubic_gradient(const Vec3f &r, float h) {
        const float PI = 3.14159265;
        const float cubicSigma = 8.f / PI / static_cast<float>(std::pow(h, 3));

        auto res = Vec3f();
        float invH = 1 / h;
        float q = r.length() * invH;

        if (q < 1e-6 || q > 1)
            return res;

        Vec3f grad_q = r / (r.length() * h);
        if (q <= 0.5)
            res = (6 * (3 * q * q - 2 * q)) * grad_q * cubicSigma;
        else {
            auto factor = 1 - q;
            res = -6 * factor * factor * grad_q * cubicSigma;
        }

        return res;
    }

    __global__ void
    computeDensity(DFSPHConstantParams *d_const,
                   DFSPHDynamicParams *d_data,
                   NeighborSearchUGConfig *d_nsConfig,
                   NeighborSearchUGParams *d_nsParams) {
        uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= d_const->particle_num)
            return;

        auto p_i = d_nsParams->particleIndices_cuData[i];

        if (d_data->mat[p_i] != FLUID)
            return;

        d_data->density[p_i] *= 0.0;
        auto pos_i = d_data->pos_adv[p_i];
        auto neib_ind = p_i * d_nsConfig->maxNeighborNum;

        for (unsigned int p_j = d_nsParams->neighbors_cuData[neib_ind], t = 0;
             p_j != UINT_MAX && t < d_nsConfig->maxNeighborNum;
             ++t, p_j = d_nsParams->neighbors_cuData[neib_ind + t]) {

            auto pos_j = d_data->pos_adv[p_j];
            d_data->density[p_i] += d_data->mass[p_j] * cubic_value(pos_i - pos_j, d_const->h);
        }

        d_data->density[p_i] = max(d_const->rest_density, d_data->density[p_i]);
        d_data->mass[p_i] = d_data->density[p_i] * d_const->rest_volume;
    }

    __global__ void
    computeDFSPHAlpha(DFSPHConstantParams *d_const,
                      DFSPHDynamicParams *d_data,
                      NeighborSearchUGConfig *d_nsConfig,
                      NeighborSearchUGParams *d_nsParams) {
        uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i >= d_const->particle_num)
            return;

        auto p_i = d_nsParams->particleIndices_cuData[i];

        if (d_data->mat[p_i] != FLUID)
            return;

        auto pos_i = d_data->pos_adv[p_i];
        auto neib_ind = p_i * d_nsConfig->maxNeighborNum;

        Vec3f alpha_1{0, 0, 0};
        float alpha_2 = 0;
        for (unsigned int p_j = d_nsParams->neighbors_cuData[neib_ind], t = 0;
             p_j != UINT_MAX && t < d_nsConfig->maxNeighborNum;
             ++t, p_j = d_nsParams->neighbors_cuData[neib_ind + t]) {

            auto pos_j = d_data->pos_adv[p_j];

            Vec3f da;
            da = d_data->mass[p_j] * cubic_gradient(pos_i - pos_j, d_const->h);
            if (d_data->mat[p_j] == FIXED_BOUND)
                da = d_data->density[p_i] * d_data->volume[p_j] * cubic_gradient(pos_i - pos_j, d_const->h);
            alpha_1 += da;
            alpha_2 += da.length() * da.length();
        }

        d_data->dfsph_alpha[p_i] = alpha_1.length() * alpha_1.length() + alpha_2 + 1e-6f;
    }

}
