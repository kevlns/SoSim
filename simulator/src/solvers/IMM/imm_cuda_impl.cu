#include "hip/hip_runtime.h"
//
// Created by ADMIN on 2024/3/26.
//
//

#include "imm_cuda_api.cuh"
#include "imm_macro.hpp"
#include "libs/SPHKernelL/kernels.cuh"
#include "libs/AnalysisL/statistic_util.hpp"

/**
 * cuda impl
 */

namespace SoSim {
    __global__ void
    init_data_cuda(IMMConstantParams *d_const,
                   IMMDynamicParams *d_data,
                   NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // TODO
        DATA_VALUE(flag_negative_vol_frac, p_i) = 0;
        DATA_VALUE(volume, p_i) = CONST_VALUE(rest_volume);
        DATA_VALUE(kappa_div, p_i) = 0;
        DATA_VALUE(color, p_i) = DATA_VALUE(vol_frac, p_i).x * CONST_VALUE(phase1_color) +
                                 DATA_VALUE(vol_frac, p_i).y * CONST_VALUE(phase2_color);
        DATA_VALUE(acc_phase_1, p_i) *= 0;
        DATA_VALUE(Cd, p_i) = CONST_VALUE(Cd0);
        DATA_VALUE(CT, p_i) = Mat33f::eye();
        DATA_VALUE(viscoelastic_stress, p_i) *= 0;
        DATA_VALUE(solution_vis, p_i) = CONST_VALUE(solution_vis_base);
        DATA_VALUE(ct_thinning_exp, p_i) = CONST_VALUE(ct_thinning_exp0);
        DATA_VALUE(ct_vis_increase_exp, p_i) = 1;
    }

    __global__ void
    update_rest_density_and_mass_cuda(IMMConstantParams *d_const,
                                      IMMDynamicParams *d_data,
                                      NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON && DATA_VALUE(mat, p_i) != Emitter_Particle)
            return;

        DATA_VALUE(rest_density, p_i) = dot(DATA_VALUE(vol_frac, p_i), CONST_VALUE(rest_density));
        DATA_VALUE(mass, p_i) = DATA_VALUE(rest_density, p_i) * DATA_VALUE(volume, p_i);
    }

    __global__ void
    update_color_cuda(IMMConstantParams *d_const,
                      IMMDynamicParams *d_data,
                      NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(color, p_i) = DATA_VALUE(vol_frac, p_i).x * CONST_VALUE(phase1_color) +
                                 DATA_VALUE(vol_frac, p_i).y * CONST_VALUE(phase2_color);
    }

    __global__ void
    recover_phase_vel_from_mixture_cuda(IMMConstantParams *d_const,
                                        IMMDynamicParams *d_data,
                                        NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON && DATA_VALUE(mat, p_i) != Emitter_Particle)
            return;

        DATA_VALUE(vel_phase_1, p_i) = DATA_VALUE(vel, p_i);
        DATA_VALUE(vel_phase_2, p_i) = DATA_VALUE(vel, p_i);
    }

    __global__ void
    compute_rigid_volume(IMMConstantParams *d_const,
                         IMMDynamicParams *d_data,
                         NeighborSearchUGConfig *d_nsConfig,
                         NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != FIXED_BOUND && DATA_VALUE(mat, p_i) != DYNAMIC_RIGID &&
            DATA_VALUE(mat, p_i) != STIR_FAN)
            return;

        auto pos_i = DATA_VALUE(pos, p_i);
        float delta = 0;
        FOR_EACH_NEIGHBOR_Pj() {
            auto pos_j = DATA_VALUE(pos, p_j);

            if (DATA_VALUE(mat, p_j) == DATA_VALUE(mat, p_i))
                delta += CUBIC_KERNEL_VALUE();
        }

        DATA_VALUE(volume, p_i) = 1.f / delta;
        DATA_VALUE(rest_density, p_i) = DATA_VALUE(volume, p_i) * CONST_VALUE(rest_bound_density);

        if (DATA_VALUE(mat, p_i) == DYNAMIC_RIGID)
            DATA_VALUE(rest_density, p_i) = DATA_VALUE(volume, p_i) * CONST_VALUE(rest_rigid_density);
    }

    __global__ void
    compute_compression_ratio_cuda(IMMConstantParams *d_const,
                                   IMMDynamicParams *d_data,
                                   NeighborSearchUGConfig *d_nsConfig,
                                   NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        DATA_VALUE(compression_ratio, p_i) *= 0;

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        auto pos_i = DATA_VALUE(pos, p_i);
        FOR_EACH_NEIGHBOR_Pj() {
            if (DATA_VALUE(mat, p_j) == Emitter_Particle)
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);

            DATA_VALUE(compression_ratio, p_i) += DATA_VALUE(volume, p_j) * CUBIC_KERNEL_VALUE();
        }
    }

    __global__ void
    compute_df_beta_cuda(IMMConstantParams *d_const,
                         IMMDynamicParams *d_data,
                         NeighborSearchUGConfig *d_nsConfig,
                         NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        DATA_VALUE(df_alpha_1, p_i) *= 0;
        DATA_VALUE(df_alpha_2, p_i) = 1e-6;

        auto pos_i = DATA_VALUE(pos, p_i);
        FOR_EACH_NEIGHBOR_Pj() {
            if (p_j == p_i || DATA_VALUE(mat, p_j) == Emitter_Particle)
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);
            auto wGrad = CUBIC_KERNEL_GRAD();

            // applied to all dynamic objects
            if (DATA_VALUE(mat, p_i) == COMMON_NEWTON)
                DATA_VALUE(df_alpha_1, p_i) += DATA_VALUE(volume, p_j) * CUBIC_KERNEL_GRAD();

            // applied to all dynamic objects
            if (DATA_VALUE(mat, p_j) == COMMON_NEWTON)
                DATA_VALUE(df_alpha_2, p_i) += dot(wGrad, wGrad) * DATA_VALUE(volume, p_j) * DATA_VALUE(volume, p_j)
                                               / DATA_VALUE(mass, p_j);
        }

        DATA_VALUE(df_alpha, p_i) =
                dot(DATA_VALUE(df_alpha_1, p_i), DATA_VALUE(df_alpha_1, p_i)) / DATA_VALUE(mass, p_i)
                + DATA_VALUE(df_alpha_2, p_i);

        if (DATA_VALUE(df_alpha, p_i) < 1e-6)
            DATA_VALUE(df_alpha, p_i) = 1e-6;
    }

    __global__ void
    compute_delta_compression_ratio_cuda(IMMConstantParams *d_const,
                                         IMMDynamicParams *d_data,
                                         NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(delta_compression_ratio, p_i) = DATA_VALUE(compression_ratio, p_i) - 1.f;
    }

    __global__ void
    update_delta_compression_ratio_from_vel_adv_cuda(IMMConstantParams *d_const,
                                                     IMMDynamicParams *d_data,
                                                     NeighborSearchUGConfig *d_nsConfig,
                                                     NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        auto pos_i = DATA_VALUE(pos, p_i);
        auto vel_adv_i = DATA_VALUE(vel_adv, p_i);
        FOR_EACH_NEIGHBOR_Pj() {
            if (p_j == p_i || DATA_VALUE(mat, p_j) == Emitter_Particle)
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);
            auto vel_adv_j = DATA_VALUE(vel_adv, p_j);
            auto wGrad = CUBIC_KERNEL_GRAD();

            DATA_VALUE(delta_compression_ratio, p_i) += dot(wGrad, vel_adv_i - vel_adv_j) *
                                                        DATA_VALUE(volume, p_j) * CONST_VALUE(dt);
        }

        if (DATA_VALUE(delta_compression_ratio, p_i) < 0)
            DATA_VALUE(delta_compression_ratio, p_i) = 0;
    }

    __global__ void
    compute_kappa_div_from_delta_compression_ratio_cuda(IMMConstantParams *d_const,
                                                        IMMDynamicParams *d_data,
                                                        NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        DATA_VALUE(kappa_div, p_i) *= 0;

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(kappa_div, p_i) = DATA_VALUE(delta_compression_ratio, p_i) / DATA_VALUE(df_alpha, p_i) *
                                     CONST_VALUE(inv_dt2) / DATA_VALUE(volume, p_i);
        DATA_VALUE(df_alpha_2, p_i) += DATA_VALUE(kappa_div, p_i);
    }

    __global__ void
    vf_update_vel_adv_from_kappa_div_cuda(IMMConstantParams *d_const,
                                          IMMDynamicParams *d_data,
                                          NeighborSearchUGConfig *d_nsConfig,
                                          NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        auto pos_i = DATA_VALUE(pos, p_i);
        FOR_EACH_NEIGHBOR_Pj() {
            if (p_j == p_i || DATA_VALUE(mat, p_j) == Emitter_Particle)
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);
            auto wGrad = CUBIC_KERNEL_GRAD();

            DATA_VALUE(vel_adv, p_i) -= CONST_VALUE(dt) * DATA_VALUE(volume, p_i) * DATA_VALUE(volume, p_j) /
                                        DATA_VALUE(mass, p_i) *
                                        (DATA_VALUE(kappa_div, p_i) + DATA_VALUE(kappa_div, p_j)) * wGrad;
        }
    }

    __global__ void
    log_kappa_div_cuda(IMMConstantParams *d_const,
                       IMMDynamicParams *d_data,
                       NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(kappa_div, p_i) = DATA_VALUE(df_alpha_2, p_i);
    }

    __global__ void
    clear_phase_acc_cuda(IMMConstantParams *d_const,
                         IMMDynamicParams *d_data,
                         NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        DATA_VALUE(acc_phase_1, p_i) *= 0;
        DATA_VALUE(acc_phase_2, p_i) *= 0;
    }

    __global__ void
    add_phase_acc_gravity_cuda(IMMConstantParams *d_const,
                               IMMDynamicParams *d_data,
                               NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(acc_phase_1, p_i) += CONST_VALUE(gravity);
        DATA_VALUE(acc_phase_2, p_i) += CONST_VALUE(gravity);
    }

    __global__ void
    add_phase_acc_vis_cuda(IMMConstantParams *d_const,
                           IMMDynamicParams *d_data,
                           NeighborSearchUGConfig *d_nsConfig,
                           NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        Vec3f acc = {0, 0, 0};
        float h2_001 = 0.001f * pow(d_const->sph_h, 2);
        auto pos_i = DATA_VALUE(pos, p_i);
        auto vel_i = DATA_VALUE(vel, p_i);
        FOR_EACH_NEIGHBOR_Pj() {
            if (DATA_VALUE(mat, p_j) != DATA_VALUE(mat, p_i) || p_j == p_i)
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);
            auto x_ij = pos_i - pos_j;
            auto vel_j = DATA_VALUE(vel, p_j);
            auto v_ij = vel_i - vel_j;
            auto wGrad = CUBIC_KERNEL_GRAD();
            auto mass_j = DATA_VALUE(mass, p_j);
            auto vis = CONST_VALUE(rest_viscosity);

            auto pi = -vis * min(0.f, dot(v_ij, pos_i - pos_j)) /
                      (x_ij.length() * x_ij.length() + h2_001);

            acc += -DATA_VALUE(mass, p_i) * mass_j * pi * wGrad;

            if (DATA_VALUE(mat, p_j) == DATA_VALUE(mat, p_i)) {
                auto v_k1_mj = DATA_VALUE(vel_phase_1, p_i) - vel_j;
                DATA_VALUE(acc_phase_1, p_i) += 10 * DATA_VALUE(volume, p_j) *
                                                dot(CONST_VALUE(phase1_vis) * (1 - CONST_VALUE(Cd)) * v_k1_mj +
                                                    (CONST_VALUE(phase1_vis) * CONST_VALUE(Cd) * v_ij),
                                                    x_ij) * wGrad / dot(x_ij, x_ij);

                auto v_k2_mj = DATA_VALUE(vel_phase_2, p_i) - vel_j;
                DATA_VALUE(acc_phase_2, p_i) += 10 * DATA_VALUE(volume, p_j) *
                                                dot(CONST_VALUE(phase2_vis) * (1 - CONST_VALUE(Cd)) * v_k2_mj +
                                                    (CONST_VALUE(phase2_vis) * CONST_VALUE(Cd) * v_ij),
                                                    x_ij) * wGrad / dot(x_ij, x_ij);
            }
        }
    }

    __global__ void
    add_phase_acc_dfsph_vis_cuda(IMMConstantParams *d_const,
                                 IMMDynamicParams *d_data,
                                 NeighborSearchUGConfig *d_nsConfig,
                                 NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        Vec3f acc = {0, 0, 0};
        float h2_001 = 0.001f * pow(CONST_VALUE(sph_h), 2);
        auto pos_i = DATA_VALUE(pos, p_i);
        auto vel_i = DATA_VALUE(vel, p_i);
        auto vis_i = CONST_VALUE(phase1_vis) * DATA_VALUE(vol_frac, p_i).x +
                     CONST_VALUE(phase2_vis) * DATA_VALUE(vol_frac, p_i).y;
        FOR_EACH_NEIGHBOR_Pj() {
            if (DATA_VALUE(mat, p_j) != DATA_VALUE(mat, p_i))
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);
            auto x_ij = pos_i - pos_j;
            auto vel_j = DATA_VALUE(vel, p_j);
            auto v_ij = vel_i - vel_j;
            auto vis_j = CONST_VALUE(phase1_vis) * DATA_VALUE(vol_frac, p_j).x +
                         CONST_VALUE(phase2_vis) * DATA_VALUE(vol_frac, p_j).y;

            auto vis = (vis_i + vis_j) / 2;

            auto pi = vis * DATA_VALUE(mass, p_j) / DATA_VALUE(rest_density, p_j) * dot(v_ij, x_ij) /
                      (x_ij.length() * x_ij.length() + h2_001);

            acc += 10 * pi * CUBIC_KERNEL_GRAD();

        }

        DATA_VALUE(acc_phase_1, p_i) += acc;
        DATA_VALUE(acc_phase_2, p_i) += acc;
    }

    __global__ void
    compute_surface_normal_cuda(IMMConstantParams *d_const,
                                IMMDynamicParams *d_data,
                                NeighborSearchUGConfig *d_nsConfig,
                                NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        auto pos_i = DATA_VALUE(pos, p_i);
        Vec3f normal;
        FOR_EACH_NEIGHBOR_Pj() {
            if (DATA_VALUE(mat, p_j) != DATA_VALUE(mat, p_i))
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);

            normal += CONST_VALUE(sph_h) * DATA_VALUE(mass, p_j) / DATA_VALUE(rest_density, p_j) *
                      cubic_gradient(pos_i - pos_j, CONST_VALUE(sph_h));
        }

        DATA_VALUE(surface_normal, p_i) = normal;
    }

    __global__ void
    add_phase_acc_surface_tension_cuda(IMMConstantParams *d_const,
                                       IMMDynamicParams *d_data,
                                       NeighborSearchUGConfig *d_nsConfig,
                                       NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        Vec3f acc = {0, 0, 0};
        float gamma = 0.005;
        auto pos_i = DATA_VALUE(pos, p_i);
        FOR_EACH_NEIGHBOR_Pj() {
            if (DATA_VALUE(mat, p_j) != DATA_VALUE(mat, p_i) || p_j == p_i)
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);
            auto k =
                    2 * DATA_VALUE(rest_density, p_i) / (DATA_VALUE(rest_density, p_i) + DATA_VALUE(rest_density, p_j));

            auto acc_1 = -gamma * DATA_VALUE(mass, p_i) * DATA_VALUE(mass, p_j) *
                         surface_tension_C((pos_i - pos_j).length(), CONST_VALUE(sph_h)) * (pos_i - pos_j) /
                         (pos_i - pos_j).length();
            auto acc_2 = -gamma * DATA_VALUE(mass, p_i) *
                         (DATA_VALUE(surface_normal, p_i) - DATA_VALUE(surface_normal, p_j));

            acc += k * (acc_1 + acc_2);
        }

        DATA_VALUE(acc_phase_1, p_i) += acc;
        DATA_VALUE(acc_phase_2, p_i) += acc;
    }

    __global__ void
    add_phase_acc_adhesion_cuda(IMMConstantParams *d_const,
                                IMMDynamicParams *d_data,
                                NeighborSearchUGConfig *d_nsConfig,
                                NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        Vec3f acc = {0, 0, 0};
        float beta = 0.0005;
        auto pos_i = DATA_VALUE(pos, p_i);
        FOR_EACH_NEIGHBOR_Pj() {
            if (DATA_VALUE(mat, p_j) != FIXED_BOUND || p_j == p_i)
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);
            auto x_ij = pos_i - pos_j;

            acc += -beta * DATA_VALUE(mass, p_i) * DATA_VALUE(volume, p_j) * adhesion_kernel_value(x_ij,
                                                                                                   CONST_VALUE(sph_h)) *
                   x_ij / x_ij.length();
        }

        DATA_VALUE(acc_phase_1, p_i) += acc;
        DATA_VALUE(acc_phase_2, p_i) += acc;
    }

    __global__ void
    correct_phase_vel_by_vis_bound_cuda(IMMConstantParams *d_const,
                                        IMMDynamicParams *d_data,
                                        NeighborSearchUGConfig *d_nsConfig,
                                        NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        int cnt = 0;
        FOR_EACH_NEIGHBOR_Pj() {
            if (DATA_VALUE(mat, p_j) != FIXED_BOUND || p_j == p_i)
                continue;

            cnt++;
        }

        float f1 = 1.f;
        if (cnt > 80)
            f1 = 0.05;

        DATA_VALUE(vel_phase_1, p_i) = f1 * DATA_VALUE(vel_phase_1, p_i);
        DATA_VALUE(vel_phase_2, p_i) = f1 * DATA_VALUE(vel_phase_2, p_i);
    }

    __global__ void
    phase_acc_2_phase_vel_cuda(IMMConstantParams *d_const,
                               IMMDynamicParams *d_data,
                               NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(vel_phase_1, p_i) += DATA_VALUE(acc_phase_1, p_i) * CONST_VALUE(dt);
        DATA_VALUE(vel_phase_2, p_i) += DATA_VALUE(acc_phase_2, p_i) * CONST_VALUE(dt);
    }

    __global__ void
    update_vel_from_phase_vel_cuda(IMMConstantParams *d_const,
                                   IMMDynamicParams *d_data,
                                   NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(vel, p_i) = DATA_VALUE(vel_phase_1, p_i) * DATA_VALUE(vol_frac, p_i).x
                               + DATA_VALUE(vel_phase_2, p_i) * DATA_VALUE(vol_frac, p_i).y;
        DATA_VALUE(vel_adv, p_i) = DATA_VALUE(vel, p_i);

        DATA_VALUE(vel_drift_phase_1, p_i) = DATA_VALUE(vel_phase_1, p_i) - DATA_VALUE(vel, p_i);
        DATA_VALUE(vel_drift_phase_2, p_i) = DATA_VALUE(vel_phase_2, p_i) - DATA_VALUE(vel, p_i);
    }

    __global__ void
    get_acc_pressure_cuda(IMMConstantParams *d_const,
                          IMMDynamicParams *d_data,
                          NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(acc, p_i) = (DATA_VALUE(vel_adv, p_i) - DATA_VALUE(vel, p_i)) * CONST_VALUE(inv_dt);
    }

    __global__ void
    distribute_acc_pressure_2_phase_cuda(IMMConstantParams *d_const,
                                         IMMDynamicParams *d_data,
                                         NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(acc_phase_1, p_i) += DATA_VALUE(acc, p_i) * (DATA_VALUE(Cd, p_i) + (1 - DATA_VALUE(Cd, p_i)) *
                                                                                      (DATA_VALUE(rest_density, p_i) /
                                                                                       CONST_VALUE(rest_density).x));
        DATA_VALUE(acc_phase_2, p_i) += DATA_VALUE(acc, p_i) * (DATA_VALUE(Cd, p_i) + (1 - DATA_VALUE(Cd, p_i)) *
                                                                                      (DATA_VALUE(rest_density, p_i) /
                                                                                       CONST_VALUE(rest_density).y));
    }

    __global__ void
    compute_kappa_incomp_from_delta_compression_ratio_cuda(IMMConstantParams *d_const,
                                                           IMMDynamicParams *d_data,
                                                           NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(kappa_incomp, p_i) = DATA_VALUE(delta_compression_ratio, p_i) / DATA_VALUE(df_alpha, p_i) *
                                        CONST_VALUE(inv_dt2) / DATA_VALUE(volume, p_i);
        DATA_VALUE(df_alpha_2, p_i) += DATA_VALUE(kappa_incomp, p_i);
    }

    __global__ void
    vf_update_vel_adv_from_kappa_incomp_cuda(IMMConstantParams *d_const,
                                             IMMDynamicParams *d_data,
                                             NeighborSearchUGConfig *d_nsConfig,
                                             NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        auto pos_i = DATA_VALUE(pos, p_i);
        FOR_EACH_NEIGHBOR_Pj() {
            if (p_j == p_i || DATA_VALUE(mat, p_j) == Emitter_Particle)
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);
            auto wGrad = CUBIC_KERNEL_GRAD();

            DATA_VALUE(vel_adv, p_i) -= CONST_VALUE(dt) * DATA_VALUE(volume, p_i) * DATA_VALUE(volume, p_j) /
                                        DATA_VALUE(mass, p_i) *
                                        (DATA_VALUE(kappa_incomp, p_i) + DATA_VALUE(kappa_incomp, p_j)) * wGrad;
        }
    }

    __global__ void
    update_pos_cuda(IMMConstantParams *d_const,
                    IMMDynamicParams *d_data,
                    NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(pos, p_i) += DATA_VALUE(vel, p_i) * CONST_VALUE(dt);
        DATA_VALUE(pos_adv, p_i) = DATA_VALUE(pos, p_i);
    }

    __global__ void
    clear_val_frac_tmp_cuda(IMMConstantParams *d_const,
                            IMMDynamicParams *d_data,
                            NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(vol_frac_in, p_i) *= 0;
        DATA_VALUE(vol_frac_out, p_i) *= 0;
    }

    __global__ void
    update_phase_change_from_drift_cuda(IMMConstantParams *d_const,
                                        IMMDynamicParams *d_data,
                                        NeighborSearchUGConfig *d_nsConfig,
                                        NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        if (DATA_VALUE(flag_negative_vol_frac, p_i) != 0)
            return;

        auto pos_i = DATA_VALUE(pos, p_i);
        FOR_EACH_NEIGHBOR_Pj() {
            if (p_j == p_i || DATA_VALUE(mat, p_j) != DATA_VALUE(mat, p_i) ||
                DATA_VALUE(flag_negative_vol_frac, p_j) != 0)
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);
            auto wGrad = CUBIC_KERNEL_GRAD();

            float vol_frac_change_1 = -CONST_VALUE(dt) * DATA_VALUE(volume, p_j) * dot(DATA_VALUE(vol_frac, p_i).x *
                                                                                       DATA_VALUE(vel_drift_phase_1,
                                                                                                  p_i) +
                                                                                       DATA_VALUE(vol_frac, p_j).x *
                                                                                       DATA_VALUE(vel_drift_phase_1,
                                                                                                  p_j),
                                                                                       wGrad);
            float vol_frac_change_2 = -CONST_VALUE(dt) * DATA_VALUE(volume, p_j) * dot(DATA_VALUE(vol_frac, p_i).y *
                                                                                       DATA_VALUE(vel_drift_phase_2,
                                                                                                  p_i) +
                                                                                       DATA_VALUE(vol_frac, p_j).y *
                                                                                       DATA_VALUE(vel_drift_phase_2,
                                                                                                  p_j),
                                                                                       wGrad);
            if (vol_frac_change_1 < 0)
                DATA_VALUE(vol_frac_out, p_i).x += vol_frac_change_1;
            else
                DATA_VALUE(vol_frac_in, p_i).x += vol_frac_change_1;

            if (vol_frac_change_2 < 0)
                DATA_VALUE(vol_frac_out, p_i).y += vol_frac_change_2;
            else
                DATA_VALUE(vol_frac_in, p_i).y += vol_frac_change_2;
        }
    }

    __global__ void
    update_phase_change_from_diffuse_cuda(IMMConstantParams *d_const,
                                          IMMDynamicParams *d_data,
                                          NeighborSearchUGConfig *d_nsConfig,
                                          NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        if (DATA_VALUE(flag_negative_vol_frac, p_i) != 0)
            return;

        auto pos_i = DATA_VALUE(pos, p_i);
        FOR_EACH_NEIGHBOR_Pj() {
            if (p_j == p_i || DATA_VALUE(mat, p_j) != DATA_VALUE(mat, p_i) ||
                DATA_VALUE(flag_negative_vol_frac, p_j) != 0)
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);
            auto x_ij = pos_i - pos_j;
            auto wGrad = CUBIC_KERNEL_GRAD();
            auto factor = dot(wGrad, x_ij) / dot(x_ij, x_ij);

            float vol_frac_ij_1 = DATA_VALUE(vol_frac, p_i).x - DATA_VALUE(vol_frac, p_j).x;
            float vol_frac_change_1 = CONST_VALUE(dt) * CONST_VALUE(Cf) * vol_frac_ij_1 * DATA_VALUE(volume, p_j) *
                                      factor;

            float vol_frac_ij_2 = DATA_VALUE(vol_frac, p_i).y - DATA_VALUE(vol_frac, p_j).y;
            float vol_frac_change_2 = CONST_VALUE(dt) * CONST_VALUE(Cf) * vol_frac_ij_2 * DATA_VALUE(volume, p_j) *
                                      factor;

            if (vol_frac_change_1 < 0)
                DATA_VALUE(vol_frac_out, p_i).x += vol_frac_change_1;
            else
                DATA_VALUE(vol_frac_in, p_i).x += vol_frac_change_1;

            if (vol_frac_change_2 < 0)
                DATA_VALUE(vol_frac_out, p_i).y += vol_frac_change_2;
            else
                DATA_VALUE(vol_frac_in, p_i).y += vol_frac_change_2;
        }
    }

    __device__ float g_all_positive;

    __global__ void
    check_negative_cuda(IMMConstantParams *d_const,
                        IMMDynamicParams *d_data,
                        NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (p_i == 0)
            g_all_positive = 1;
        __syncthreads();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        if (DATA_VALUE(flag_negative_vol_frac, p_i) != 0)
            return;

        auto vol_frac_tmp = DATA_VALUE(vol_frac, p_i) + DATA_VALUE(vol_frac_out, p_i) + DATA_VALUE(vol_frac_in, p_i);
        if (vol_frac_tmp.x < 0 || vol_frac_tmp.y < 0) {
            DATA_VALUE(flag_negative_vol_frac, p_i) = 1;
            atomicAdd(&g_all_positive, -1);
        }
    }

    __global__ void
    update_phase_change_cuda(IMMConstantParams *d_const,
                             IMMDynamicParams *d_data,
                             NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(vol_frac, p_i) += (DATA_VALUE(vol_frac_in, p_i) + DATA_VALUE(vol_frac_out, p_i));
    }

    __global__ void
    release_unused_drift_vel_cuda(IMMConstantParams *d_const,
                                  IMMDynamicParams *d_data,
                                  NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        if (DATA_VALUE(flag_negative_vol_frac, p_i) != 0) {
            DATA_VALUE(vel_phase_1, p_i) = DATA_VALUE(vel, p_i);
            DATA_VALUE(vel_phase_2, p_i) = DATA_VALUE(vel, p_i);
        }
    }

    __global__ void
    release_negative_cuda(IMMConstantParams *d_const,
                          IMMDynamicParams *d_data,
                          NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(flag_negative_vol_frac, p_i) = 0;
    }

    __global__ void
    regularize_val_frac_cuda(IMMConstantParams *d_const,
                             IMMDynamicParams *d_data,
                             NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        float frac_sum = DATA_VALUE(vol_frac, p_i).x + DATA_VALUE(vol_frac, p_i).y;
        DATA_VALUE(vol_frac, p_i) /= frac_sum;
    }

    __global__ void
    estimate_density_cuda(IMMConstantParams *d_const,
                          IMMDynamicParams *d_data,
                          NeighborSearchUGConfig *d_nsConfig,
                          NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        DATA_VALUE(density_sph, p_i) *= 0;
        auto pos_i = DATA_VALUE(pos, p_i);
        auto rest_dens_i = DATA_VALUE(rest_density, p_i);

        FOR_EACH_NEIGHBOR_Pj() {
            auto pos_j = DATA_VALUE(pos, p_j);
            auto rest_dens_j = DATA_VALUE(rest_density, p_j);
            if (DATA_VALUE(mat, p_j) != COMMON_NEWTON)
                rest_dens_j = rest_dens_i;

            DATA_VALUE(density_sph, p_i) += rest_dens_j * CONST_VALUE(rest_volume) * CUBIC_KERNEL_VALUE();
        }

        DATA_VALUE(density_sph, p_i) = fmax(DATA_VALUE(density_sph, p_i), rest_dens_i);
    }

    __global__ void
    compute_vel_grad_cuda(IMMConstantParams *d_const,
                          IMMDynamicParams *d_data,
                          NeighborSearchUGConfig *d_nsConfig,
                          NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        // applied to all dynamic objects
        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        // TODO
        auto pos_i = DATA_VALUE(pos, p_i);
        auto vel_i = DATA_VALUE(vel, p_i);
        Mat33f vGrad_sum;

        FOR_EACH_NEIGHBOR_Pj() {

            if (DATA_VALUE(mat, p_j) != COMMON_NEWTON)
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);
            auto vel_j = DATA_VALUE(vel, p_j);
            auto vel_ji = vel_j - vel_i;
            auto wGrad = CUBIC_KERNEL_GRAD();
            auto volume_j = DATA_VALUE(volume, p_j);

            vGrad_sum += volume_j * vel_ji * wGrad;
        }

        DATA_VALUE(vel_grad, p_i) = vGrad_sum;
    }

    __global__ void
    update_conformation_tensor_cuda(IMMConstantParams *d_const,
                                    IMMDynamicParams *d_data,
                                    NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        auto CT_i = DATA_VALUE(CT, p_i);

        Mat33f dQ = (CT_i * DATA_VALUE(vel_grad, p_i) +
                     DATA_VALUE(vel_grad, p_i).transpose() * CT_i - 1.f /
                                                                    (CONST_VALUE(ct_relaxation_time) +
                                                                     1e-5f) *
                                                                    (CT_i -
                                                                     Mat33f::eye())) * CONST_VALUE(dt) -
                    DATA_VALUE(ct_thinning_exp, p_i) * (CT_i - Mat33f::eye()) * CT_i;
        DATA_VALUE(CT, p_i) += dQ;
        DATA_VALUE(solution_vis, p_i) = DATA_VALUE(ct_vis_increase_exp, p_i) * CONST_VALUE(solution_vis_base);
        DATA_VALUE(solution_vis, p_i) = fmin(DATA_VALUE(solution_vis, p_i), CONST_VALUE(solution_vis_max));

        auto D = 0.5 * (DATA_VALUE(vel_grad, p_i) + DATA_VALUE(vel_grad, p_i).transpose());
        auto shearRate = sqrtf(0.5f * D.trace() * D.trace());
        DATA_VALUE(solution_vis, p_i) =
                DATA_VALUE(solution_vis, p_i) + (CONST_VALUE(solution_vis_max) - DATA_VALUE(solution_vis, p_i)) /
                                                (1 + pow(10 * shearRate, 5)) * fmin(1.f, DATA_VALUE(vol_frac, p_i).y /
                                                                                         CONST_VALUE(
                                                                                                 polymer_vol_frac0));

        DATA_VALUE(viscoelastic_stress, p_i) =
                DATA_VALUE(solution_vis, p_i) * (DATA_VALUE(CT, p_i) - Mat33f::eye());
    }

    __global__ void
    add_viscoelastic_acc_cuda(IMMConstantParams *d_const,
                              IMMDynamicParams *d_data,
                              NeighborSearchUGConfig *d_nsConfig,
                              NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        auto pos_i = DATA_VALUE(pos, p_i);
        auto dens_i = DATA_VALUE(density_sph, p_i);
        Vec3f acc;

        FOR_EACH_NEIGHBOR_Pj() {

            if (DATA_VALUE(mat, p_j) != COMMON_NEWTON)
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);
            auto wGrad = CUBIC_KERNEL_GRAD();
            auto dens_j = DATA_VALUE(density_sph, p_j);

            acc += (DATA_VALUE(viscoelastic_stress, p_i) / powf(dens_i, 2) +
                    DATA_VALUE(viscoelastic_stress, p_j) / powf(dens_j, 2)) * wGrad;
        }

        DATA_VALUE(acc_phase_1, p_i) += acc * DATA_VALUE(rest_density, p_i) * DATA_VALUE(vol_frac, p_i).y;
        DATA_VALUE(acc_phase_2, p_i) += acc * DATA_VALUE(rest_density, p_i) * DATA_VALUE(vol_frac, p_i).y;
    }

    __global__ void
    update_Cd_cuda(IMMConstantParams *d_const,
                   IMMDynamicParams *d_data,
                   NeighborSearchUGConfig *d_nsConfig,
                   NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        auto pos_i = DATA_VALUE(pos, p_i);
        DATA_VALUE(blocking_factor, p_i) *= 0;
        FOR_EACH_NEIGHBOR_Pj() {
            if (DATA_VALUE(mat, p_j) != DATA_VALUE(mat, p_i) || p_i == p_j)
                continue;

            auto pos_j = DATA_VALUE(pos, p_j);

            DATA_VALUE(blocking_factor, p_i) +=
                    DATA_VALUE(volume, p_j) * DATA_VALUE(vol_frac, p_j).y * CUBIC_KERNEL_VALUE();
        }

        DATA_VALUE(Cd, p_i) = CONST_VALUE(Cd0) + (1 - CONST_VALUE(Cd0)) * DATA_VALUE(blocking_factor, p_i);
    }

    __global__ void
    compute_shear_exp_cuda(IMMConstantParams *d_const,
                           IMMDynamicParams *d_data,
                           NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        // thinning coefficient: beta
        DATA_VALUE(ct_thinning_exp, p_i) = CONST_VALUE(ct_thinning_exp0) * fmax(0.f, CONST_VALUE(polymer_vol_frac0) -
                                                                                     DATA_VALUE(vol_frac, p_i).y) /
                                           (CONST_VALUE(polymer_vol_frac0) + 1e-6);

        // vis increase coefficient: kappa
        DATA_VALUE(ct_vis_increase_exp, p_i) = powf(1.f - DATA_VALUE(vol_frac, p_i).y, -1.2);
    }

}

namespace SoSim { // extra func cuda impl
    __global__ void
    stir_fan_cuda(IMMConstantParams *d_const,
                  IMMDynamicParams *d_data,
                  NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != STIR_FAN)
            return;

        const float M_PI = 3.1415926;
        float angleRadians = -0.004f * (M_PI / 180.0f);// 将角度转换为弧度
        float cosAngle = cos(angleRadians);
        float sinAngle = sin(angleRadians);

        Vec3f center_offset = {0, 0, 0};

        auto pos = DATA_VALUE(pos, p_i) - center_offset;
        DATA_VALUE(pos, p_i).x = pos.x * cosAngle - pos.z * sinAngle + center_offset.x;
        DATA_VALUE(pos, p_i).z = pos.x * sinAngle + pos.z * cosAngle + center_offset.z;
    }

    __global__ void
    buckling_move_cuda(IMMConstantParams *d_const,
                       IMMDynamicParams *d_data,
                       NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != MOVING_TUBE && DATA_VALUE(mat, p_i) != MOVING_COVER)
            return;

        DATA_VALUE(pos, p_i) += CONST_VALUE(dt) * DATA_VALUE(vel, p_i);
    }

    __global__ void
    rotate_bowl_cuda(IMMConstantParams *d_const,
                     IMMDynamicParams *d_data,
                     NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != MOVING_BOWL && DATA_VALUE(mat, p_i) != STIR_FAN)
            return;

        const float M_PI = 3.1415926;
        float angleRadians = -0.0001;// 将角度转换为弧度
        float cosAngle = cos(angleRadians);
        float sinAngle = sin(angleRadians);

        Vec3f offset = {0, -6.8165, 0};

        auto pos = DATA_VALUE(pos, p_i) - offset;
        DATA_VALUE(pos, p_i).y = pos.y * cosAngle - pos.z * sinAngle + offset.y;
        DATA_VALUE(pos, p_i).z = pos.y * sinAngle + pos.z * cosAngle + offset.z;
    }

    __global__ void
    correct_phase_vel_by_artificial_vis_bound_cuda(IMMConstantParams *d_const,
                                                   IMMDynamicParams *d_data,
                                                   NeighborSearchUGConfig *d_nsConfig,
                                                   NeighborSearchUGParams *d_nsParams) {
        CHECK_THREAD();

        if (DATA_VALUE(mat, p_i) != COMMON_NEWTON)
            return;

        int cnt = 0;
        FOR_EACH_NEIGHBOR_Pj() {
            if (DATA_VALUE(mat, p_j) == DATA_VALUE(mat, p_i) || DATA_VALUE(mat, p_j) == Emitter_Particle)
                continue;

            cnt++;
        }

        float f1 = 1;
        if (cnt > 15)
            f1 = 1;

        DATA_VALUE(vel_phase_1, p_i) *= f1;
        DATA_VALUE(vel_phase_2, p_i) *= f1;
    }
}


/**
 * host invoke impl
 */

namespace SoSim {
    __host__ void
    init_data(IMMConstantParams &h_const,
              IMMConstantParams *d_const,
              IMMDynamicParams *d_data,
              NeighborSearchUGParams *d_nsParams) {
        init_data_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);
    }

    __host__ void
    prepare_ims(IMMConstantParams &h_const,
                IMMConstantParams *d_const,
                IMMDynamicParams *d_data,
                NeighborSearchUGConfig *d_nsConfig,
                NeighborSearchUGParams *d_nsParams) {
        // ims update_rest_density_and_mass()
        update_rest_density_and_mass_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // ims recover_phase_vel_from_mixture()
        recover_phase_vel_from_mixture_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // compute_rigid_volume()
        compute_rigid_volume<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);
    }

    __host__ void
    sph_precompute(IMMConstantParams &h_const,
                   IMMConstantParams *d_const,
                   IMMDynamicParams *d_data,
                   NeighborSearchUGConfig *d_nsConfig,
                   NeighborSearchUGParams *d_nsParams) {
        // compute_compression_ratio(), AKA step_sph_compute_compression_ratio()
        compute_compression_ratio_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);

        // compute_df_beta(), AKA step_df_compute_beta()
        compute_df_beta_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);
    }

    __host__ void
    vfsph_div(IMMConstantParams &h_const,
              IMMDynamicParams &h_data,
              Vec3ui &obj_part_index,
              IMMConstantParams *d_const,
              IMMDynamicParams *d_data,
              NeighborSearchUGConfig *d_nsConfig,
              NeighborSearchUGParams *d_nsParams,
              bool &crash) {
        int iter = 0;
        while (true) {
            iter++;

            // compute_delta_compression_ratio()
            compute_delta_compression_ratio_cuda<<<h_const.block_num, h_const.thread_num>>>(
                    d_const, d_data, d_nsParams);

            std::vector<float> de(h_const.particle_num);
            hipMemcpy(de.data(), h_data.delta_compression_ratio, h_const.particle_num * sizeof(float),
                       hipMemcpyDeviceToHost);

            // update_delta_compression_ratio_from_vel_adv()
            update_delta_compression_ratio_from_vel_adv_cuda<<<h_const.block_num, h_const.thread_num>>>(
                    d_const, d_data, d_nsConfig, d_nsParams);

            hipMemcpy(de.data(), h_data.delta_compression_ratio, h_const.particle_num * sizeof(float),
                       hipMemcpyDeviceToHost);

            // update_vf_compressible_ratio()
            auto compressible_ratio = cal_mean(h_data.delta_compression_ratio,
                                               h_const.particle_num, obj_part_index.y);

            // compute_kappa_div_from_delta_compression_ratio()
            compute_kappa_div_from_delta_compression_ratio_cuda<<<h_const.block_num, h_const.thread_num>>>(
                    d_const, d_data, d_nsParams);

            std::vector<float> kd(h_const.particle_num);
            hipMemcpy(kd.data(), h_data.kappa_div, h_const.particle_num * sizeof(float),
                       hipMemcpyDeviceToHost);

            std::vector<float> mass(h_const.particle_num);
            hipMemcpy(mass.data(), h_data.mass, h_const.particle_num * sizeof(float),
                       hipMemcpyDeviceToHost);

            // vf_update_vel_adv_from_kappa_div()
            vf_update_vel_adv_from_kappa_div_cuda<<<h_const.block_num, h_const.thread_num>>>(
                    d_const, d_data, d_nsConfig, d_nsParams);

            // check compressible_ratio
            if (compressible_ratio < h_const.div_free_threshold || iter > 100)
                break;
        }

        // log_kappa_div()
//        log_kappa_div_cuda<<<h_const.block_num, h_const.thread_num>>>(
//                d_const, d_data, d_nsParams);

        std::cout << "div-iter: " << iter << '\n';

        if (iter == 101)
            crash = true;

        // vel = vel_adv
        //        hipMemcpy(h_data.vel, h_data.vel_adv, h_const.particle_num * sizeof(Vec3f), hipMemcpyDeviceToDevice);
    }

    __host__ void
    apply_pressure_acc(IMMConstantParams &h_const,
                       IMMConstantParams *d_const,
                       IMMDynamicParams *d_data,
                       NeighborSearchUGParams *d_nsParams) {
        // get_acc_pressure()
        get_acc_pressure_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // clear_phase_acc()
        clear_phase_acc_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // distribute_acc_pressure_2_phase()
        distribute_acc_pressure_2_phase_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // phase_acc_2_phase_vel()
        phase_acc_2_phase_vel_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // update_vel_from_phase_vel()
        update_vel_from_phase_vel_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);
    }

    __host__ void
    ism_gravity_vis_surface(IMMConstantParams &h_const,
                            IMMConstantParams *d_const,
                            IMMDynamicParams *d_data,
                            NeighborSearchUGConfig *d_nsConfig,
                            NeighborSearchUGParams *d_nsParams) {
        // clear_phase_acc()
        clear_phase_acc_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // add_phase_acc_gravity()
        add_phase_acc_gravity_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // add_phase_acc_vis()
        add_phase_acc_vis_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);

        // compute_surface_normal
        compute_surface_normal_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);

        // add_phase_acc_surface_tension_cuda()
        add_phase_acc_surface_tension_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);

        // phase_acc_2_phase_vel()
        phase_acc_2_phase_vel_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // update_vel_from_phase_vel()
        update_vel_from_phase_vel_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);
    }

    __host__ void
    dfsph_gravity_vis_surface(IMMConstantParams &h_const,
                              IMMConstantParams *d_const,
                              IMMDynamicParams *d_data,
                              NeighborSearchUGConfig *d_nsConfig,
                              NeighborSearchUGParams *d_nsParams) {
        // clear_phase_acc()
        clear_phase_acc_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // add_phase_acc_gravity()
        add_phase_acc_gravity_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // add_phase_acc_vis()
        add_phase_acc_dfsph_vis_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);

        // compute_surface_normal
        compute_surface_normal_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);

        // add_phase_acc_surface_tension_cuda()
        add_phase_acc_surface_tension_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);

        // phase_acc_2_phase_vel()
        phase_acc_2_phase_vel_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // update_vel_from_phase_vel()
        update_vel_from_phase_vel_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);
    }

    __host__ void
    vfsph_incomp(IMMConstantParams &h_const,
                 IMMDynamicParams &h_data,
                 Vec3ui &obj_part_index,
                 IMMConstantParams *d_const,
                 IMMDynamicParams *d_data,
                 NeighborSearchUGConfig *d_nsConfig,
                 NeighborSearchUGParams *d_nsParams,
                 bool &crash) {
        int iter = 0;
        while (true) {
            iter++;

            // compute_delta_compression_ratio()
            compute_delta_compression_ratio_cuda<<<h_const.block_num, h_const.thread_num>>>(
                    d_const, d_data, d_nsParams);

            // update_delta_compression_ratio_from_vel_adv()
            update_delta_compression_ratio_from_vel_adv_cuda<<<h_const.block_num, h_const.thread_num>>>(
                    d_const, d_data, d_nsConfig, d_nsParams);

            // update_vf_compressible_ratio()
            auto compressible_ratio = cal_mean(h_data.delta_compression_ratio,
                                               h_const.particle_num, obj_part_index.y);

            // compute_kappa_incomp_from_delta_compression_ratio()
            compute_kappa_incomp_from_delta_compression_ratio_cuda<<<h_const.block_num, h_const.thread_num>>>(
                    d_const, d_data, d_nsParams);

            // vf_update_vel_adv_from_kappa_incomp()
            vf_update_vel_adv_from_kappa_incomp_cuda<<<h_const.block_num, h_const.thread_num>>>(
                    d_const, d_data, d_nsConfig, d_nsParams);

            // check compressible_ratio
            if (compressible_ratio < h_const.incompressible_threshold || iter > 100)
                break;
        }

        std::cout << "incomp-iter: " << iter << '\n';

        if (iter == 101)
            crash = true;
    }

    __host__ void
    update_pos(IMMConstantParams &h_const,
               IMMConstantParams *d_const,
               IMMDynamicParams *d_data,
               NeighborSearchUGParams *d_nsParams) {
        update_pos_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);
    }

    __host__ void
    phase_transport_ism(IMMConstantParams &h_const,
                        IMMConstantParams *d_const,
                        IMMDynamicParams *d_data,
                        NeighborSearchUGConfig *d_nsConfig,
                        NeighborSearchUGParams *d_nsParams,
                        bool &crash) {
        // clear_val_frac_tmp()
        clear_val_frac_tmp_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // update_phase_change_from_drift()
        update_phase_change_from_drift_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);

        // update_phase_change_from_diffuse()
        update_phase_change_from_diffuse_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);

        // while: check_negative(), update_phase_change_from_drift(), update_phase_change_from_diffuse()
        float all_positive = 0;
        int iter = 1;
        while (true) {
            // check
            check_negative_cuda<<<h_const.block_num, h_const.thread_num>>>(
                    d_const, d_data, d_nsParams);
            hipMemcpyFromSymbol(&all_positive, HIP_SYMBOL(g_all_positive), sizeof(float), 0, hipMemcpyDeviceToHost);
            if (all_positive == 1 || iter > 100)
                break;

            // clear_val_frac_tmp()
            clear_val_frac_tmp_cuda<<<h_const.block_num, h_const.thread_num>>>(
                    d_const, d_data, d_nsParams);

            // update_phase_change_from_drift()
            update_phase_change_from_drift_cuda<<<h_const.block_num, h_const.thread_num>>>(
                    d_const, d_data, d_nsConfig, d_nsParams);

            // update_phase_change_from_diffuse()
            update_phase_change_from_diffuse_cuda<<<h_const.block_num, h_const.thread_num>>>(
                    d_const, d_data, d_nsConfig, d_nsParams);

            iter++;
        }

        std::cout << "phase-trans-iter: " << iter << "\n";

        if (iter == 101)
            crash = true;

        // update_phase_change()
        update_phase_change_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // release_unused_drift_vel()
        release_unused_drift_vel_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // release_negative()
        release_negative_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // regularize_val_frac()
        regularize_val_frac_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // update_rest_density_and_mass()
        update_rest_density_and_mass_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // update_vel_from_phase_vel()
        update_vel_from_phase_vel_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);
    }

    __host__ void
    update_mass_and_vel(IMMConstantParams &h_const,
                        IMMConstantParams *d_const,
                        IMMDynamicParams *d_data,
                        NeighborSearchUGParams *d_nsParams) {
        // regularize_val_frac()
        regularize_val_frac_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // update_rest_density_and_mass()
        update_rest_density_and_mass_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // update_vel_from_phase_vel()
        update_vel_from_phase_vel_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);
    }

    __host__ void
    update_color(IMMConstantParams &h_const,
                 IMMConstantParams *d_const,
                 IMMDynamicParams *d_data,
                 NeighborSearchUGParams *d_nsParams) {
        // get_acc_pressure()
        update_color_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);
    }

    __host__ void
    ism_viscoelastic(IMMConstantParams &h_const,
                     IMMConstantParams *d_const,
                     IMMDynamicParams *d_data,
                     NeighborSearchUGConfig *d_nsConfig,
                     NeighborSearchUGParams *d_nsParams) {
        // estimate_density()
        estimate_density_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);

        // compute_vel_grad()
        compute_vel_grad_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);

        // update_conformation_tensor()
        update_conformation_tensor_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // clear_phase_acc()
        clear_phase_acc_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        add_viscoelastic_acc_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);

        // phase_acc_2_phase_vel()
        phase_acc_2_phase_vel_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);

        // update_vel_from_phase_vel()
        update_vel_from_phase_vel_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);
    }

    __host__ void
    update_CT_parameters(IMMConstantParams &h_const,
                         IMMConstantParams *d_const,
                         IMMDynamicParams *d_data,
                         NeighborSearchUGConfig *d_nsConfig,
                         NeighborSearchUGParams *d_nsParams) {
        // set_Cd()
//        update_Cd_cuda<<<h_const.block_num, h_const.thread_num>>>(
//                d_const, d_data, d_nsConfig, d_nsParams);

        // compute_shear_exp()
        compute_shear_exp_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);
    }

    __host__ void
    artificial_vis_bound(IMMConstantParams &h_const,
                         IMMConstantParams *d_const,
                         IMMDynamicParams *d_data,
                         NeighborSearchUGConfig *d_nsConfig,
                         NeighborSearchUGParams *d_nsParams) {
        // correct_phase_vel_by_artificial_vis_bound()
        correct_phase_vel_by_artificial_vis_bound_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsConfig, d_nsParams);

        // update_vel_from_phase_vel()
        update_vel_from_phase_vel_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);
    }
}

namespace SoSim { // extra func host invoke
    __host__ void
    stirring(IMMConstantParams &h_const,
             IMMConstantParams *d_const,
             IMMDynamicParams *d_data,
             NeighborSearchUGParams *d_nsParams) {
        stir_fan_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);
    }

    __host__ void
    rotate_bowl(IMMConstantParams &h_const,
                IMMConstantParams *d_const,
                IMMDynamicParams *d_data,
                NeighborSearchUGParams *d_nsParams) {
        rotate_bowl_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);
    }

    __host__ void
    buckling(IMMConstantParams &h_const,
             IMMConstantParams *d_const,
             IMMDynamicParams *d_data,
             NeighborSearchUGParams *d_nsParams) {
        buckling_move_cuda<<<h_const.block_num, h_const.thread_num>>>(
                d_const, d_data, d_nsParams);
    }
}